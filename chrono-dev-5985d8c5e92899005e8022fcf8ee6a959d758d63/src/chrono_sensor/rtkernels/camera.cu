#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for tracing and measureing depth for a typical pinhole camera
//
// =============================================================================

#include <hip/hip_math_constants.h>
#include <optixu/optixu_aabb.h>
#include "chrono_sensor/rtkernels/ray_utils.h"

using namespace optix;

rtDeclareVariable(PerRayData_camera, prd_camera, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, max_scene_distance, , );
rtDeclareVariable(rtObject, root_node, , );
rtDeclareVariable(float3, default_color, , );
rtDeclareVariable(float, default_depth, , );
rtDeclareVariable(int, max_depth, , );

// camera parameters
rtDeclareVariable(float3, c_pos, , );      // camera position
rtDeclareVariable(float3, c_up, , );       // camera up vector
rtDeclareVariable(float3, c_forward, , );  // camera forward vector
rtDeclareVariable(float3, c_left, , );     // camera left vector
rtDeclareVariable(float, c_hFOV, , );      // camera horizontal field of view
// rtDeclareVariable(float, c_vFOV, , );      // camera vertical field of view

// environment map
rtTextureSampler<float4, 2> environment_map;
rtDeclareVariable(int, has_environment_map, , );

rtBuffer<uchar4, 2> output_buffer;  // byte version

// This kernel is launched once for each pixel in the image
RT_PROGRAM void pinhole_camera() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;
    d.y *= (float)(screen.y) / (float)(screen.x);
    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float h_factor = c_hFOV / HIP_PI_F * 2.0;
    // float v_factor = c_vFOV / HIP_PI_F * 2.0;
    float3 ray_direction = normalize(c_forward - d.x * c_left * h_factor + d.y * c_up * h_factor);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, CAMERA_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_camera prd_camera = make_camera_data(make_float3(0), 1.f, 1);

    // launch the ray
    rtTrace(root_node, ray, prd_camera, RT_RAY_FLAG_DISABLE_ANYHIT);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = make_color(prd_camera.color);
}

// This kernel is launched once for each pixel in the image
RT_PROGRAM void fov_lens_camera() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;
    // rescale y direction to be proportional to x
    d.y *= (float)(screen.y) / (float)(screen.x);

    if (abs(d.x) > 1e-5 || abs(d.y) > 1e-5) {
        // float omega = c_hFOV * (tan(tan(c_hFOV / 2.0)) / tan(c_hFOV / 2.0));

        float r1 = sqrtf(d.x * d.x + d.y * d.y);

        // float r2 = tan(A * r1 * c_hFOV / 2.0) / (tan(c_hFOV / 2.0));
        float r2 = tan(r1 * tan(c_hFOV / 2.0)) / tan(c_hFOV / 2.0);
        // float r2 = r1 / cos(c_hFOV / 2);

        float scaled_extent = tan(tan(c_hFOV / 2.0)) / tan(c_hFOV / 2.0);

        // float y_max = 1.4142;

        d.x = d.x * (r2 / r1) / scaled_extent;
        d.y = d.y * (r2 / r1) / scaled_extent;
    }

    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float h_factor = c_hFOV / HIP_PI_F * 2.0;
    // float v_factor = c_vFOV / HIP_PI_F * 2.0;
    float3 ray_direction = normalize(c_forward - d.x * c_left * h_factor + d.y * c_up * h_factor);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, CAMERA_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_camera prd_camera = make_camera_data(make_float3(0), 1.f, 1);

    // launch the ray
    rtTrace(root_node, ray, prd_camera, RT_RAY_FLAG_DISABLE_ANYHIT);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = make_color(prd_camera.color);
}

RT_PROGRAM void thinlens_camera() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;
    d.y *= (float)(screen.y) / (float)(screen.x);
    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float h_factor = c_hFOV / HIP_PI_F * 2.0;
    // float v_factor = c_vFOV / HIP_PI_F * 2.0;
    float3 ray_direction = normalize(c_forward - d.x * c_left * h_factor + d.y * c_up * h_factor);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, CAMERA_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_camera prd_camera = make_camera_data(make_float3(0), 1.f, 1);

    // launch the ray
    rtTrace(root_node, ray, prd_camera, RT_RAY_FLAG_DISABLE_ANYHIT);

    // TODO: this should be defined in the camera sensor class
    float dir_sensor_len_ratio = .5f;  // ratio of direction incident on sensor to direction incident on lens
    float3 sensor_incident_vec = normalize(dir_sensor_len_ratio * ray_direction + c_forward);

    float3 distort_color = prd_camera.color * powf((dot(c_forward, sensor_incident_vec)), 4);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = make_color(distort_color);
}
