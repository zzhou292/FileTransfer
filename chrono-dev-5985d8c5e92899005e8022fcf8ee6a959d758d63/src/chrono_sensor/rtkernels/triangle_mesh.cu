#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for mesh geometries
//
// =============================================================================

#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

rtBuffer<int3> index_buffer;
rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;

RT_PROGRAM void mesh_attributes() {
    const int3 vertex_idx = index_buffer[rtGetPrimitiveIndex()];
    const float2 bary_coord = rtGetTriangleBarycentrics();

    if (normal_buffer.size() > 0) {
        shading_normal =
            normalize(normal_buffer[vertex_idx.y] * bary_coord.x + normal_buffer[vertex_idx.z] * bary_coord.y +
                      normal_buffer[vertex_idx.x] * (1.0f - bary_coord.x - bary_coord.y));
    } else {
        shading_normal = normalize(cross(vertex_buffer[vertex_idx.y] - vertex_buffer[vertex_idx.x],
                                         vertex_buffer[vertex_idx.z] - vertex_buffer[vertex_idx.x]));
    }
    if (texcoord_buffer.size() == 0) {
        texcoord = make_float2(0.0f);
    } else {
        texcoord = texcoord_buffer[vertex_idx.y] * bary_coord.x + texcoord_buffer[vertex_idx.z] * bary_coord.y +
                   texcoord_buffer[vertex_idx.x] * (1.0f - bary_coord.x - bary_coord.y);
    }
}
