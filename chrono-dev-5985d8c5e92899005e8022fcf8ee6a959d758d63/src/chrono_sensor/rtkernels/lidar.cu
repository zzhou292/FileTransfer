#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for tracing and measureing depth for a typical pinhole camera
//
// =============================================================================

#include <hip/hip_math_constants.h>
#include <optixu/optixu_aabb.h>
#include "chrono_sensor/rtkernels/ray_utils.h"

using namespace optix;

rtDeclareVariable(PerRayData_lidar, prd_lidar, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, max_scene_distance, , );
rtDeclareVariable(rtObject, root_node, , );

// camera parameters
rtDeclareVariable(float3, c_pos, , );      // lidar position
rtDeclareVariable(float3, c_up, , );       // lidar up vector
rtDeclareVariable(float3, c_forward, , );  // lidar forward vector
rtDeclareVariable(float3, c_left, , );     // lidar left vector
rtDeclareVariable(float, c_hFOV, , );      // lidar horizontal field of view
rtDeclareVariable(float, c_vFOV, , );      // lidar vertical field of view

rtBuffer<float2, 2> output_buffer;  // byte version

// for beam divergence and multi-sample kernel
rtDeclareVariable(float, divergence_angle, , );  // lidar beam divergence
rtDeclareVariable(int, ray_samples, , );         // samples per beam

// This kernel is launched once for each pixel in the image
RT_PROGRAM void spherical() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;  //[-1,1]

    // float ranges[ray_samples];
    // float intensities[ray_samples];

    // float for (int i = 0; i < ray_samples; i++) {
    float theta = d.x * c_hFOV / 2.0;
    float phi = d.y * c_vFOV / 2.0;
    float xy_proj = cos(phi);

    float z = sin(phi);
    float y = xy_proj * sin(theta);
    float x = xy_proj * cos(theta);

    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float3 ray_direction = normalize(c_forward * x + c_left * y + c_up * z);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, LIDAR_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_lidar prd_lidar = make_lidar_data(0, 1.f, 0);

    // launch the ray
    rtTrace(root_node, ray, prd_lidar,RT_RAY_FLAG_DISABLE_ANYHIT);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = make_float2(prd_lidar.range, prd_lidar.intensity);
}

RT_PROGRAM void multi_sample() {
    float div_angle = divergence_angle;
    int sample_radius = ray_samples;

    size_t2 screen = output_buffer.size();
    int2 global_beam_dims = make_int2(screen.x / (sample_radius * 2 - 1), screen.y / (sample_radius * 2 - 1));
    int2 local_beam_dims = make_int2(sample_radius * 2 - 1, sample_radius * 2 - 1);

    // index of center of beam
    int beam_index_x = launch_index.x / (sample_radius * 2 - 1);
    int beam_index_y = launch_index.y / (sample_radius * 2 - 1);
    float2 beam_id_fraction =
        (make_float2(beam_index_x, beam_index_y) + make_float2(0.5, 0.5)) / make_float2(global_beam_dims) * 2.f -
        1.f;  //[-1,1]

    // theta and phi for beam center
    float beam_theta = beam_id_fraction.x * c_hFOV / 2.0;
    float beam_phi = beam_id_fraction.y * c_vFOV / 2.0;

    // index of local ray in beam
    int local_ray_index_x = launch_index.x % (sample_radius * 2 - 1);
    int local_ray_index_y = launch_index.y % (sample_radius * 2 - 1);
    float2 local_ray_id_fraction = (make_float2(local_ray_index_x, local_ray_index_y) + make_float2(0.5, 0.5)) /
                                       make_float2(local_beam_dims) * 2.f -
                                   1.f;  //[-1,1]

    // relative theta and phi for local ray in beam
    float local_ray_theta = local_ray_id_fraction.x * div_angle / 2.0;
    float local_ray_phi = local_ray_id_fraction.y * div_angle / 2.0;

    // carry on ray-tracing per ray
    float theta = beam_theta + local_ray_theta;
    float phi = beam_phi + local_ray_phi;

    float xy_proj = cos(phi);

    float z = sin(phi);
    float y = xy_proj * sin(theta);
    float x = xy_proj * cos(theta);

    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float3 ray_direction = normalize(c_forward * x + c_left * y + c_up * z);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, LIDAR_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_lidar prd_lidar = make_lidar_data(0, 1.f, 0);

    // launch the ray
    rtTrace(root_node, ray, prd_lidar,RT_RAY_FLAG_DISABLE_ANYHIT);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = make_float2(prd_lidar.range, prd_lidar.intensity);
}
