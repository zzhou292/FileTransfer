// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace chrono {
namespace sensor {

__global__ void init_random_states(unsigned int seed, hiprandState_t* rng_states, int n_generators) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_generators) {
        hiprand_init(seed, index, 0, &rng_states[index]);
    }
}
}  // namespace sensor
}  // namespace chrono
