#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Eric Brandt, Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "curand_utils.cuh"
#include "camera_noise.cuh"
#include "chrono_sensor/utils/CudaMallocHelper.h"

#include <chrono>
#include <memory>

namespace chrono {
namespace sensor {

// cuda_camera_noise_pixel_dependent
bool init_camera_noise_const_normal = true;
std::shared_ptr<hiprandState_t> camera_noise_const_normal_rng;

// cuda_camera_noise_pixel_dependent
bool init_camera_noise_pix_dep = true;
std::shared_ptr<hiprandState_t> camera_noise_pix_dep_rng;

// cuda_camera_noise_crf
bool init_camera_noise_crf = true;
std::shared_ptr<hiprandState_t> camera_noise_crf_rng;

// Add random normal noise to the image with constant mean and stdev
__global__ void const_normal_noise_kernel(unsigned char* bufPtr,
                                          int w,
                                          int h,
                                          float mean,
                                          float stdev,
                                          hiprandState_t* rng_states) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < w * h) {
        // hiprand_normal(&rng_states[index]);
        float r_rand = hiprand_normal(&rng_states[index]) * stdev + mean;
        float g_rand = hiprand_normal(&rng_states[index]) * stdev + mean;
        float b_rand = hiprand_normal(&rng_states[index]) * stdev + mean;

        // get pixel values
        unsigned char pix_r = bufPtr[index * 4];
        unsigned char pix_g = bufPtr[index * 4 + 1];
        unsigned char pix_b = bufPtr[index * 4 + 2];

        // convert to float and add noise
        float r = ((float)(pix_r)) / 255.0 + r_rand;
        float g = ((float)(pix_g)) / 255.0 + g_rand;
        float b = ((float)(pix_b)) / 255.0 + b_rand;

        // convert back to char and save in image
        bufPtr[index * 4] = (unsigned char)(r * 255.999);
        bufPtr[index * 4 + 1] = (unsigned char)(g * 255.999);
        bufPtr[index * 4 + 2] = (unsigned char)(b * 255.999);
    }
}

__global__ void pix_dep_noise_kernel(unsigned char* bufPtr,
                                     int w,
                                     int h,
                                     float gain,
                                     float sigma_shot,
                                     float sigma_adc,
                                     hiprandState_t* rng_states) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < w * h) {
        // get pixel values
        unsigned char pix_r = bufPtr[index * 4];
        unsigned char pix_g = bufPtr[index * 4 + 1];
        unsigned char pix_b = bufPtr[index * 4 + 2];

        float r = ((float)(pix_r)) / 255.0;
        float g = ((float)(pix_g)) / 255.0;
        float b = ((float)(pix_b)) / 255.0;

        // hiprand_normal(&rng_states[index]);
        float stdev_r = sqrtf((r * sigma_shot * sigma_shot) + (sigma_adc * sigma_adc));
        float stdev_g = sqrtf((g * sigma_shot * sigma_shot) + (sigma_adc * sigma_adc));
        float stdev_b = sqrtf((b * sigma_shot * sigma_shot) + (sigma_adc * sigma_adc));
        float r_rand = hiprand_normal(&rng_states[index]) * stdev_r;
        float g_rand = hiprand_normal(&rng_states[index]) * stdev_g;
        float b_rand = hiprand_normal(&rng_states[index]) * stdev_b;

        // convert to float and add noise
        r = r + r_rand;
        g = g + g_rand;
        b = b + b_rand;

        // convert back to char and save in image
        bufPtr[index * 4] = (unsigned char)(r * 255.999);
        bufPtr[index * 4 + 1] = (unsigned char)(g * 255.999);
        bufPtr[index * 4 + 2] = (unsigned char)(b * 255.999);
    }
}

void cuda_camera_noise_const_normal(void* bufPtr, int width, int height, float mean, float stdev) {
    const int nThreads = 512;
    int nBlocks = (width * height + nThreads - 1) / nThreads;

    if (init_camera_noise_const_normal) {
        camera_noise_const_normal_rng = std::shared_ptr<hiprandState_t>(
            cudaMallocHelper<hiprandState_t>(nBlocks * nThreads), cudaFreeHelper<hiprandState_t>);
        init_camera_noise_const_normal = false;
        init_random_states<<<nBlocks, nThreads>>>(std::chrono::high_resolution_clock::now().time_since_epoch().count(),
                                                  camera_noise_const_normal_rng.get(), nBlocks * nThreads);
    }

    const_normal_noise_kernel<<<nBlocks, nThreads>>>((unsigned char*)bufPtr, width, height, mean, stdev,
                                                     camera_noise_const_normal_rng.get());
}

void cuda_camera_noise_pixel_dependent(void* bufPtr,
                                       int width,
                                       int height,
                                       float gain,
                                       float sigma_read,
                                       float sigma_adc) {
    const int nThreads = 512;
    int nBlocks = (width * height + nThreads - 1) / nThreads;

    if (init_camera_noise_pix_dep) {
        camera_noise_pix_dep_rng = std::shared_ptr<hiprandState_t>(cudaMallocHelper<hiprandState_t>(nBlocks * nThreads),
                                                                  cudaFreeHelper<hiprandState_t>);
        init_camera_noise_pix_dep = false;
        init_random_states<<<nBlocks, nThreads>>>(std::chrono::high_resolution_clock::now().time_since_epoch().count(),
                                                  camera_noise_pix_dep_rng.get(), nBlocks * nThreads);
    }

    pix_dep_noise_kernel<<<nBlocks, nThreads>>>((unsigned char*)bufPtr, width, height, gain, sigma_read, sigma_adc,
                                                camera_noise_pix_dep_rng.get());
}

}  // namespace sensor
}  // namespace chrono
